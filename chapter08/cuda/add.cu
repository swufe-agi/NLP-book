
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <time.h>

#define N 100000000

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void) {
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = rand() / (float)RAND_MAX;
    y[i] = rand() / (float)RAND_MAX;
  }

  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC, &start);
  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &end);
  double duration =
      (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
  printf("Time taken: %f seconds\n", duration);

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
